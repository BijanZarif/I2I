#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/vision_layers.hpp"

namespace caffe {

// Copy (one line per thread) from one array to another, with arbitrary
// strides in the last all dimensions.
template <typename Dtype>
__global__ void copy_kernel_forward(const int num_lines, 
    const int line_size, const int num_axes,
    const int* top_shape_data, const int* bottom_shape_data,
    const int* crop_data, const Dtype* bottom, Dtype* top) {
  CUDA_KERNEL_LOOP(line, num_lines) {
    int k = line;
    int top_index = 0;
    int bottom_index = crop_data[num_axes-1];
    int top_offset = top_shape_data[num_axes-1];
    int bottom_offset = bottom_shape_data[num_axes-1];
    // Calcuate top and bottom index
    for (int j = num_axes - 2; j >= 0; --j) {
      int topsub = k % top_shape_data[j];
      top_index += topsub*top_offset;
      top_offset *= top_shape_data[j];

      int bottomsub = topsub+crop_data[j];
      bottom_index += bottomsub*bottom_offset;
      bottom_offset *= bottom_shape_data[j];
      k /= top_shape_data[j];
    }
    // Copy line bottom -> top
    for (int i = 0; i < line_size; ++i) {
      top[top_index + i] = bottom[bottom_index + i];
    }
  }
}

template <typename Dtype>
__global__ void copy_kernel_backward(const int num_lines,
    const int line_size, const int num_axes,
    const int* top_shape_data,const int* bottom_shape_data,
    const int* crop_data, const Dtype* top, Dtype* bottom) {
  CUDA_KERNEL_LOOP(line, num_lines) {
    int k = line;
    int top_index = 0;
    int bottom_index = crop_data[num_axes-1];
    int top_offset = top_shape_data[num_axes-1];
    int bottom_offset = bottom_shape_data[num_axes-1];
    // Calcuate top and bottom index
    for (int j = num_axes - 2; j >= 0; --j) {
      int topsub = k % top_shape_data[j];
      top_index += topsub*top_offset;
      top_offset *= top_shape_data[j];

      int bottomsub = topsub+crop_data[j];
      bottom_index += bottomsub*bottom_offset;
      bottom_offset *= bottom_shape_data[j];
      k /= top_shape_data[j];
    }
    // Copy line top_diff -> bottom_diff
    for (int i = 0; i < line_size; ++i) {
      bottom[bottom_index + i] = top[top_index + i];
    }
    
  }
}

template <typename Dtype>
void CropLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  //const int* crop_data = crop_.cpu_data();
  //for (int i = 0; i < num_axes_-2;++i)
    //printf("%d, ",crop_data[i]);
  //printf("\n");

  copy_kernel_forward<<<CAFFE_GET_BLOCKS(num_lines_), CAFFE_CUDA_NUM_THREADS>>>(
      num_lines_, line_size_, num_axes_,
      top_shape_.gpu_data(), bottom_shape_.gpu_data(),
      crop_.gpu_data(), bottom_data, top_data);
}

template <typename Dtype>
void CropLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  if (propagate_down[0]) {
    caffe_gpu_set(bottom[0]->count(), static_cast<Dtype>(0), bottom_diff);
    // NOLINT_NEXT_LINE(whitespace/operators)
    copy_kernel_backward<<<CAFFE_GET_BLOCKS(num_lines_), CAFFE_CUDA_NUM_THREADS>>>(
      num_lines_, line_size_, num_axes_,
      top_shape_.gpu_data(), bottom_shape_.gpu_data(),
      crop_.gpu_data(), top_diff, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CropLayer);

}  // namespace caffe
